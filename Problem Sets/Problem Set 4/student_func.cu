#include "hip/hip_runtime.h"
//Udacity HW 4
//Radix Sorting

#include "utils.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>





/* Red Eye Removal
   ===============

   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   ==========================================

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

   1) Histogram of the number of occurrences of each digit
   2) Exclusive Prefix Sum of Histogram
   3) Determine relative offset of each digit
        For example [0 0 1 1 0 0 1]
                ->  [0 1 0 1 2 3 2]
   4) Combine the results of steps 2 & 3 to determine the final
      output location for each element and move it there

   LSB Radix sort is an out-of-place sort and you will need to ping-pong values
   between the input and output buffers we have provided.  Make sure the final
   sorted results end up in the output buffer!  Hint: You may need to do a copy
   at the end.

 */

// Creates a histogram using atomicAdd. Could be done faster, alot faster
__global__
void histogramKernel(unsigned int* d_bins, unsigned int* d_inputs,
                    int size,
                    unsigned int k){
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if(tid >= size){
        return;
    }
    int binId = (d_inputs[tid] & (1 << k)) >> k;
    atomicAdd(&d_bins[binId], 1);
}



void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               const size_t numElems)
{
  //TODO
  //PUT YOUR SORT HERE
  // THrust ftw lol
  thrust::device_ptr<unsigned int> d_inputV(d_inputVals);
  thrust::device_ptr<unsigned int> d_inputP(d_inputPos);
  thrust::sort_by_key(d_inputV, d_inputV + numElems, d_inputPos);


}
